#ifndef GPU_RAY_H
#define GPU_RAY_H
#include "hip/hip_math_constants.h"
#include "gpuVector3D.cu"
struct gpuRay {
  size_t depth;

  gpuVector3D o;
  gpuVector3D d;
  mutable double min_t;
  mutable double max_t;

  gpuVector3D inv_d;
  int sign[3];

  /* Constructors */
  __device__
    gpuRay(const gpuVector3D& o, const gpuVector3D& d, int depth = 0)
      : o(o), d(d), min_t(0.0), depth(depth) {
    inv_d = gpuVector3D(1 / d.x, 1 / d.y, 1 / d.z);
    sign[0] = (inv_d.x < 0);
    sign[1] = (inv_d.y < 0);
    sign[2] = (inv_d.z < 0);
    max_t = HIP_INF;
  }

  __device__
    gpuRay(const gpuVector3D& o, const gpuVector3D& d, double max_t, int depth = 0)
      : o(o), d(d), min_t(0.0), max_t(max_t), depth(depth) {
    inv_d = gpuVector3D(1 / d.x, 1 / d.y, 1 / d.z);
    sign[0] = (inv_d.x < 0);
    sign[1] = (inv_d.y < 0);
    sign[2] = (inv_d.z < 0);
  }

  __device__
    inline gpuVector3D at_time(double t) const { return o + t * d; }

  /*
  gpuRay transform_by(const gpuMatrix4x4& t) const {
    const gpuVector4D& new0 = t * gpuVector4D(o, 1.0);
    return gpuRay((new0 / new0.w).to3D(), (t * gpuVector4D(d, 0.0)).to3D());
    
  }*/
};

struct LoggedRay {

  __device__
  LoggedRay(const gpuRay& r, double hit_t)
    : o(r.o), d(r.d), hit_t(hit_t) {}

  gpuVector3D o;
  gpuVector3D d;
  double hit_t;
};

#endif
