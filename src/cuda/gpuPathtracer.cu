#include "hip/hip_runtime.h"
#include "gpuPathtracer.h"
#include "../static_scene/triangle.h"
#include "../static_scene/object.h"

using namespace CMU462;
using namespace StaticScene;

__global__ void render()
{
}

gpuPathTracer::gpuPathTracer(PathTracer *__pathtracer)
{
  pathtracer = __pathtracer;
}

void gpuPathTracer::load_scene()
{
  // using the CPU's bvh, load the mesh information
  size_t num_tri = pathtracer->bvh->primitives.size();
  const Mesh* cpu_mesh = ((Triangle*)(pathtracer->bvh->primitives[0]))->mesh;
  size_t numVerts = cpu_mesh->numVerts;

  // Copy over the vertices and normals of the mesh
  gpuVector3D *pos_d;
  gpuVector3D *norm_d;

  hipMalloc((void**)&pos_d,sizeof(gpuVector3D) * numVerts);
  hipMalloc((void**)&norm_d,sizeof(gpuVector3D) * numVerts);

  hipMemcpy(pos_d, cpu_mesh->positions, sizeof(gpuVector3D) * numVerts,hipMemcpyHostToDevice);
  hipMemcpy(norm_d, cpu_mesh->normals, sizeof(gpuVector3D) * numVerts,hipMemcpyHostToDevice);

  // Group the mesh info into a gpuMesh
  gpuMesh gpu_mesh(pos_d,norm_d);

  hipMalloc((void**)&mesh,sizeof(gpuMesh));

  hipMemcpy(mesh,&gpu_mesh, sizeof(gpuMesh),hipMemcpyHostToDevice);

  // Copy over the triangles
  gpuTriangle* temp_tri = new gpuTriangle[num_tri];
  for(int i = 0; i < num_tri; i++)
  {
    temp_tri[i] = gpuTriangle(mesh,
        ((Triangle*)(pathtracer->bvh->primitives[i]))->v1,
        ((Triangle*)(pathtracer->bvh->primitives[i]))->v2,
        ((Triangle*)(pathtracer->bvh->primitives[i]))->v3);
  }

  hipMalloc((void**)&primitives,sizeof(gpuTriangle) * num_tri);
  hipMemcpy(primitives,temp_tri,sizeof(gpuTriangle) * num_tri,hipMemcpyHostToDevice);
}

void gpuPathTracer::load_camera()
{
}

void gpuPathTracer::set_frame_size(size_t width, size_t height)
{
  w = width;
  h = height;

  // reallocate the imagePixels buffer
  hipFree(imagePixels);
  hipMalloc((void**)&imagePixels,sizeof(bool) * w * h);
}

void gpuPathTracer::update_screen()
{
}

void gpuPathTracer::start_raytrace()
{
}

void gpuPathTracer::build_accel()
{
}

__device__ bool gpuPathTracer::trace_ray(const gpuRay& ray)
{
}

__device__ bool gpuPathTracer::raytrace_pixel(size_t x, size_t y)
{
}


