#include "hip/hip_runtime.h"
#include "gpuPathtracer.h"
#include "../static_scene/triangle.h"
#include "../static_scene/object.h"
#include "gpuRay.cu"
#include "gpuMesh.cu"
#include "gpuTriangle.cu"
#include "gpuVector3D.cu"
#include "gpuCamera.cu"

#ifdef DEBUG
#define CHK(ans) {gpuAssert((ans), __FILE__, __LINE__);}
#define POSTKERNEL CHK(hipPeekAtLastError())
#else
#define CHK(ans)
#endif
inline void gpuAssert(hipError_t code, const char *file, int line)
{
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s %s %s\n",
        hipGetErrorString(code),file,line);
    exit(code);
  }
}
using namespace CMU462;
using namespace StaticScene;

__constant__ gpuMesh* mesh;
__constant__ gpuTriangle* primitives;
__constant__ gpuCamera* camera_const;
__constant__ bool* imagePixels_const;
__constant__ size_t w_d;
__constant__ size_t h_d;
__constant__ size_t numPrim;

bool* imagePixels;
gpuCamera* camera;
gpuTriangle* gpu_primitives;
gpuMesh *gpu_mesh;

// returns the result of ray tracing intersection with the scene primitives
__device__ bool trace_ray(const gpuRay& ray)
{
  for(size_t i = 0; i < numPrim; i++)
  {
    if(primitives[i].intersect(ray))
      return true;
  }
  return false;
}

// Using the x and y position of the pixel, create a ray and use trace_ray
__device__ bool raytrace_pixel(size_t x, size_t y)
{
  gpuVector3D p((x + 0.5)/w_d,(y + 0.5)/h_d,0);
  return trace_ray(camera_const->generate_ray(p.x,p.y));
}

// kernel for doing raytracing
__global__ void render()
{
  size_t index = blockIdx.x * blockDim.x + threadIdx.x;
  size_t x = index % w_d;
  size_t y = index / w_d;
  printf("%u\n", index);
  if(x < w_d && y < h_d)
  {
    imagePixels_const[index] = true;//raytrace_pixel(x,y);
  }
}


gpuPathTracer::gpuPathTracer(PathTracer *__pathtracer)
{
  pathtracer = __pathtracer;
  hipSetDevice(1);
}

gpuPathTracer::~gpuPathTracer() {
  hipFree(camera);
  hipFree(imagePixels);
  hipFree(gpu_primitives);
  hipFree(gpu_mesh);
}

void gpuPathTracer::load_scene()
{
  // using the CPU's bvh, load the mesh information
  size_t num_tri = pathtracer->bvh->primitives.size();
  hipMemcpyToSymbol(HIP_SYMBOL(numPrim),&num_tri,sizeof(size_t));

  const Mesh* cpu_mesh = ((Triangle*)(pathtracer->bvh->primitives[0]))->mesh;
  size_t numVerts = cpu_mesh->numVerts;

  // Copy over the vertices and normals of the mesh
  gpuVector3D *pos_d;
  gpuVector3D *norm_d;

  hipMalloc((void**)&pos_d,sizeof(gpuVector3D) * numVerts);
  hipMalloc((void**)&norm_d,sizeof(gpuVector3D) * numVerts);

  hipMemcpy(pos_d, cpu_mesh->positions, sizeof(gpuVector3D) * numVerts,
        hipMemcpyHostToDevice);
  hipMemcpy(norm_d, cpu_mesh->normals, sizeof(gpuVector3D) * numVerts,
        hipMemcpyHostToDevice);

  // Group the mesh info into a gpuMesh
  gpuMesh gpu_mesh_tmp(pos_d,norm_d);

  hipMalloc((void**)&gpu_mesh,sizeof(gpuMesh));

  hipMemcpy(gpu_mesh,&gpu_mesh_tmp, sizeof(gpuMesh),hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(mesh),&gpu_mesh,sizeof(gpuMesh*));

  // Copy over the triangles
  gpuTriangle* temp_tri = new gpuTriangle[num_tri];
  for(int i = 0; i < num_tri; i++)
  {
    temp_tri[i] = gpuTriangle(cpu_mesh,gpu_mesh,
        ((Triangle*)(pathtracer->bvh->primitives[i]))->v1,
        ((Triangle*)(pathtracer->bvh->primitives[i]))->v2,
        ((Triangle*)(pathtracer->bvh->primitives[i]))->v3);
  }
  hipMalloc((void**)&gpu_primitives,sizeof(gpuTriangle) * num_tri);
  hipMemcpy(gpu_primitives,temp_tri,sizeof(gpuTriangle) * num_tri,
        hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(primitives),&gpu_primitives,sizeof(gpuTriangle*));

  printf("[GPU Pathtracer]: finished loading scene\n");
}

void gpuPathTracer::load_camera(Camera *cam)
{
  gpuCamera temp = gpuCamera(cam->c2w, cam->position(),
      cam->screenW, cam->screenH, cam->screenDist);
  hipMalloc((void**)&camera,sizeof(gpuCamera));
  hipMemcpy(camera,&temp,sizeof(gpuCamera),hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(camera_const),&camera,sizeof(gpuCamera*),hipMemcpyHostToDevice);
}

void gpuPathTracer::set_frame_size(size_t width, size_t height)
{
  w = width;
  h = height;

  hipMemcpyToSymbol(HIP_SYMBOL(w_d),&w,sizeof(size_t));
  hipMemcpyToSymbol(HIP_SYMBOL(h_d),&h,sizeof(size_t));

  // reallocate the imagePixels buffer
  hipMalloc((void**)&imagePixels,sizeof(bool) * w * h);
  hipMemcpyToSymbol(HIP_SYMBOL(imagePixels_const),&imagePixels,sizeof(bool*));
}

// Takes the bool imagePixels and draws it on the screen as b/w pixels
void gpuPathTracer::update_screen()
{
  Color white(1, 1, 1, 1);
  Color black(0, 0, 0, 0);

  bool *tmp = new bool[w * h];
  hipMemcpy(tmp, imagePixels, w * h * sizeof(bool),
        hipMemcpyDeviceToHost);
  //copy imagePixels into pathtracer->frameBuffer
  for(size_t i = 0; i < h; i++) {
    for(size_t j = 0; j < w; j++) {
      if(tmp[i * w + j]) {
        pathtracer->frameBuffer.update_pixel(white, j, i);
      }
      else {
        pathtracer->frameBuffer.update_pixel(black, j, i);
      }
    }
  }
  delete[] tmp;
  pathtracer->doneState();
}

// Wrapper for lanching the render() kernel
void gpuPathTracer::start_raytrace()
{
  size_t numBlocks = (w * h + 31 -1)/32;
  render<<<32,32>>>();
  hipDeviceSynchronize();
  printf("[GPU Pathtracer]: finished rendering scene\n");
}

