#include "hip/hip_runtime.h"
#include "gpuPathtracer.h"
#include "../static_scene/triangle.h"
#include "../static_scene/object.h"
#include "gpuRay.cu"
#include "gpuMesh.cu"
#include "gpuTriangle.cu"
#include "gpuVector3D.cu"
#include "gpuCamera.cu"
//#include "gpuBvh.cu"
#include "gpuBBox.cu"

#ifdef DEBUG
#define CHK(ans) {gpuAssert((ans), __FILE__, __LINE__);}
#define POSTKERNEL CHK(hipPeekAtLastError())
#else
#define CHK(ans)
#endif
inline void gpuAssert(hipError_t code, const char *file, int line)
{
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s %s %s\n",
        hipGetErrorString(code),file,line);
    exit(code);
  }
}
using namespace CMU462;
using namespace StaticScene;

__constant__ gpuTriangle* primitives;
//__constant__ gpuCamera* camera_const;
__constant__ bool* imagePixels_const;
__constant__ size_t w_d;
__constant__ size_t h_d;
__constant__ size_t numPrim;
__constant__ gpuVector3D* pos;

bool* imagePixels;
gpuCamera* camera;
gpuTriangle* gpu_primitives;
gpuVector3D *pos_d;

// returns the result of ray tracing intersection with the scene primitives
__device__ bool trace_ray(gpuRay ray)
{
  for(size_t i = 0; i < numPrim; i++)
  {
    if(primitives[i].intersect(ray))
      return true;
  }
  return false;
}

// Using the x and y position of the pixel, create a ray and use trace_ray
__device__ bool raytrace_pixel(size_t x, size_t y,gpuCamera* cam)
{
  gpuVector3D p((x + 0.5)/w_d,(y + 0.5)/h_d,0);
  return trace_ray(cam->generate_ray(p.x,p.y));
}

// kernel for doing raytracing
__global__ void render(gpuCamera* cam)
{
  size_t index = blockIdx.x * blockDim.x + threadIdx.x;
  size_t x = index % w_d;
  size_t y = index / w_d;
  //printf("%u\n", index);
  if(x < w_d && y < h_d)
  {
    imagePixels_const[index] = raytrace_pixel(x,y,cam);
  }
}


gpuPathTracer::gpuPathTracer(PathTracer *__pathtracer)
{
  pathtracer = __pathtracer;
}

gpuPathTracer::~gpuPathTracer() {
  hipFree(camera);
  hipFree(imagePixels);
  hipFree(gpu_primitives);
  hipFree(pos_d);
}

void gpuPathTracer::load_scene()
{
  timer.start();
  // using the CPU's bvh, load the mesh information
  size_t num_tri = pathtracer->bvh->primitives.size();
  hipMemcpyToSymbol(HIP_SYMBOL(numPrim),&num_tri,sizeof(size_t));

  const Mesh* cpu_mesh = ((Triangle*)(pathtracer->bvh->primitives[0]))->mesh;
  size_t numVerts = cpu_mesh->numVerts;
  printf("numVerts: %d\n",numVerts);

  gpuVector3D* pos_temp = new gpuVector3D[numVerts];
  
  for(size_t i = 0; i < numVerts; i++)
  {
    pos_temp[i] = gpuVector3D(cpu_mesh->positions[i].x,
        cpu_mesh->positions[i].y,
        cpu_mesh->positions[i].z);
  }
  // Copy over the vertices and normals of the mesh

  hipMalloc((void**)&pos_d,sizeof(gpuVector3D) * numVerts);

  hipMemcpy(pos_d, pos_temp, sizeof(gpuVector3D) * numVerts,
      hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(pos),&pos_d,sizeof(gpuVector3D*));

  // Copy over the triangles
  gpuTriangle* temp_tri = new gpuTriangle[num_tri];
  for(int i = 0; i < num_tri; i++)
  {
    temp_tri[i] = gpuTriangle(cpu_mesh,pos_d,
        ((Triangle*)(pathtracer->bvh->primitives[i]))->v1,
        ((Triangle*)(pathtracer->bvh->primitives[i]))->v2,
        ((Triangle*)(pathtracer->bvh->primitives[i]))->v3);
  }
  hipMalloc((void**)&gpu_primitives,sizeof(gpuTriangle) * num_tri);
  hipMemcpy(gpu_primitives,temp_tri,sizeof(gpuTriangle) * num_tri,
      hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(primitives),&gpu_primitives,sizeof(gpuTriangle*));
  timer.stop();

  printf("[GPU Pathtracer]: finished loading scene (%.4f sec)\n",timer.duration());
}

void gpuPathTracer::load_camera(Camera *cam)
{
  gpuCamera temp = gpuCamera(cam->c2w, cam->position(),
      cam->screenW, cam->screenH, cam->screenDist);
  hipMalloc((void**)&camera,sizeof(gpuCamera));
  hipMemcpy(camera,&temp,sizeof(gpuCamera),hipMemcpyHostToDevice);
  hipMemcpy(&temp,camera,sizeof(gpuCamera),hipMemcpyDeviceToHost);
  printf("w %u, h %u, d %f\n",temp.screenW, temp.screenH, temp.screenDist);
  printf("w %u, h %u, d %f\n",cam->screenW, cam->screenH, cam->screenDist);
  //hipMemcpyToSymbol(HIP_SYMBOL(camera_const),camera,sizeof(gpuCamera*),hipMemcpyHostToDevice);
}

void gpuPathTracer::set_frame_size(size_t width, size_t height)
{
  w = width;
  h = height;

  hipMemcpyToSymbol(HIP_SYMBOL(w_d),&w,sizeof(size_t));
  hipMemcpyToSymbol(HIP_SYMBOL(h_d),&h,sizeof(size_t));

  // reallocate the imagePixels buffer
  hipMalloc((void**)&imagePixels,sizeof(bool) * w * h);
  hipMemcpyToSymbol(HIP_SYMBOL(imagePixels_const),&imagePixels,sizeof(bool*));
}

// Takes the bool imagePixels and draws it on the screen as b/w pixels
void gpuPathTracer::update_screen()
{
  Color white(1, 1, 1, 1);
  Color black(0, 0, 0, 0);

  bool *tmp = new bool[w * h];
  hipMemcpy(tmp, imagePixels, w * h * sizeof(bool),
      hipMemcpyDeviceToHost);
  //copy imagePixels into pathtracer->frameBuffer
  for(size_t i = 0; i < h; i++) {
    for(size_t j = 0; j < w; j++) {
      if(tmp[i * w + j]) {
        pathtracer->frameBuffer.update_pixel(white, j, i);
      }
      else {
        pathtracer->frameBuffer.update_pixel(black, j, i);
      }
    }
  }
  delete[] tmp;
  pathtracer->doneState();
}

// Wrapper for lanching the render() kernel
void gpuPathTracer::start_raytrace()
{
  timer.start();
  size_t numBlocks = (w * h + 31 -1)/32;
  render<<<numBlocks,32>>>(camera);
  hipDeviceSynchronize();
  timer.stop();
  printf("[GPU Pathtracer]: finished rendering scene (%.4f sec)\n",timer.duration());
}

