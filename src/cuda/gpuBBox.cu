#include "hip/hip_runtime.h"
#ifndef gpu_bbox
#define gpu_bbox

#include "gpuVector3D.cu"
#include "gpuRay.cu"
#include "hip/hip_math_constants.h"

#define MIN(X, Y) ((X) < (Y) ? (X) : (Y))
#define MAX(X, Y) ((X) > (Y) ? (X) : (Y))


struct gpuBBox {

  gpuVector3D max;     ///< min corner of the bounding box
  gpuVector3D min;     ///< max corner of the bounding box
  gpuVector3D extent;  ///< extent of the bounding box (min -> max)

  /**
   * Constructor.
   * The default constructor creates a new bounding box which contains no
   * points.
   */
  __device__ __host__ gpuBBox() {
    max = gpuVector3D(0, 0, 0);
    min = gpuVector3D( 0,  0, 0);
    extent = max - min;
  }

  /**
   * Constructor.
   * Creates a bounding box that includes a single point.
   */
  __device__ __host__
    gpuBBox(const gpuVector3D p) : min(p), max(p) { extent = max - min; }

  /**
   * Constructor.
   * Creates a bounding box with given bounds.
   * \param min the min corner
   * \param max the max corner
   */
  __device__ __host__ 
    gpuBBox(const gpuVector3D min, const gpuVector3D max) :
    min(min), max(max) { extent = max - min; }

  /**
   * Constructor.
   * Creates a bounding box with given bounds (component wise).
   */
  __device__ __host__
    gpuBBox(const float minX, const float minY, const float minZ,
      const float maxX, const float maxY, const float maxZ) {
    min = gpuVector3D(minX, minY, minZ);
    max = gpuVector3D(maxX, maxY, maxZ);
    extent = max - min;
  }

  __device__ __host__
    gpuBBox(BBox bb) {
      min = gpuVector3D(bb.min.x, bb.min.y, bb.min.z);
      max = gpuVector3D(bb.max.x, bb.max.y, bb.max.z);
      extent = max - min;
    }
  
  /**
   * Expand the bounding box to include another (union).
   * If the given bounding box is contained within *this*, nothing happens.
   * Otherwise *this* is expanded to the minimum volume that contains the
   * given input.
   * \param bbox the bounding box to be included
   */
  __device__ void expand(const gpuBBox bbox) {
    min.x = fminf(min.x, bbox.min.x);
    min.y = fminf(min.y, bbox.min.y);
    min.z = fminf(min.z, bbox.min.z);
    max.x = fmaxf(max.x, bbox.max.x);
    max.y = fmaxf(max.y, bbox.max.y);
    max.z = fmaxf(max.z, bbox.max.z);
    extent = max - min;
  }

  /**
   * Expand the bounding box to include a new point in space.
   * If the given point is already inside *this*, nothing happens.
   * Otherwise *this* is expanded to a minimum volume that contains the given
   * point.
   * \param p the point to be included
   */
  __device__ void expand(const gpuVector3D p) {
    min.x = fminf(min.x, p.x);
    min.y = fminf(min.y, p.y);
    min.z = fminf(min.z, p.z);
    max.x = fmaxf(max.x, p.x);
    max.y = fmaxf(max.y, p.y);
    max.z = fmaxf(max.z, p.z);
    extent = max - min;
  }

  __device__ gpuVector3D centroid() const {
    return (min + max) / 2;
  }

  /**
   * Compute the surface area of the bounding box.
   * \return surface area of the bounding box.
   */
  __device__ float surface_area() const {
    if (empty()) return 0.0;
    return 2 * (extent.x * extent.z +
        extent.x * extent.y +
        extent.y * extent.z);
  }

  /**
   * Check if bounding box is empty.
   * Bounding box that has no size is considered empty. Note that since
   * bounding box are used for objects with positive volumes, a bounding
   * box of zero size (empty, or contains a single vertex) are considered
   * empty.
   */
  __device__ bool empty() const {
    return min.x > max.x || min.y > max.y || min.z > max.z;
  }

  /**
   * Ray - bbox intersection.
   * Intersects ray with bounding box, does not store shading information.
   * \param r the ray to intersect with
   * \param t0 lower bound of intersection time
   * \param t1 upper bound of intersection time
   */
 __device__ bool intersect(const gpuRay r, float t0, float t1) const {
  float tx1 = (min.x - r.o.x) / r.d.x;
  float tx2 = (max.x - r.o.x) / r.d.x;
  float txmin = MIN(tx1, tx2);
  float txmax = MAX(tx1, tx2);
  
  float ty1 = (min.y - r.o.y) / r.d.y;
  float ty2 = (max.y - r.o.y) / r.d.y;
  float tymin = MIN(ty1, ty2);
  float tymax = MAX(ty1, ty2);

  float tz1 = (min.z - r.o.z) / r.d.z; 
  float tz2 = (min.z - r.o.z) / r.d.z;
  float tzmin = MIN(tz1, tz2);
  float tzmax = MAX(tz1, tz2);

  float tmin = MAX(MAX(t0, txmin), MAX(tymin, tzmin));
  float tmax = MIN(MIN(t1, txmax), MIN(tymax, tzmax));

  if(tmin <= tmax && tmax >= 0) {
    t1 = tmax;
    t0 = tmin;
    return true;
  }

  return false;
 }

};
#endif
